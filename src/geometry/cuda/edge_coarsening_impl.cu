#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file geometry/cuda/edge_coarsening_impl.cu
 * \brief Edge coarsening CUDA implementation
 */
#include <dgl/array.h>
#include <dgl/random.h>
#include <dmlc/thread_local.h>
#include <hiprand.h>
#include <cstdint>
#include "../geometry_op.h"
#include "../../runtime/cuda/cuda_common.h"
#include "../../array/cuda/utils.h"

#define BLOCKS(N, T) (N + T - 1) / T

namespace dgl {
namespace geometry {
namespace impl {

constexpr float BLUE_P = 0.53406;
constexpr int BLUE = -1;
constexpr int RED = -2;
constexpr int EMPTY_IDX = -1;

__device__ bool done_d;
__global__ void init_done_kernel() { done_d = true; }

template <typename IdType>
__global__ void colorize_kernel(const float *prop, int64_t num_elem, IdType *result) {
  const IdType idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elem) {
    if (result[idx] < 0) {  // if unmatched
      result[idx] = (prop[idx] > BLUE_P) ? RED : BLUE;
      done_d = false;
    }
  }
}

template <typename FloatType, typename IdType>
__global__ void weighted_propose_kernel(const IdType *indptr, const IdType *indices,
                                        const FloatType *weights, int64_t num_elem,
                                        IdType *proposal, IdType *result) {
  const IdType idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elem) {
    if (result[idx] != BLUE) return;

    bool has_unmatched_neighbor = false;
    FloatType weight_max = 0.;
    IdType v_max = EMPTY_IDX;

    for (IdType i = indptr[idx]; i < indptr[idx + 1]; ++i) {
      auto v = indices[i];

      if (result[v] < 0)
        has_unmatched_neighbor = true;
      if (result[v] == RED && weights[i] >= weight_max) {
        v_max = v;
        weight_max = weights[i];
      }
    }

    proposal[idx] = v_max;
    if (!has_unmatched_neighbor)
      result[idx] = idx;
  }
}

template <typename FloatType, typename IdType>
__global__ void weighted_respond_kernel(const IdType *indptr, const IdType *indices,
                                        const FloatType *weights, int64_t num_elem,
                                        IdType *proposal, IdType *result) {
  const IdType idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elem) {
    if (result[idx] != RED) return;

    bool has_unmatched_neighbors = false;
    IdType v_max = -1;
    FloatType weight_max = 0.;

    for (IdType i = indptr[idx]; i < indptr[idx + 1]; ++i) {
      auto v = indices[i];

      if (result[v] < 0) {
        has_unmatched_neighbors = true;
      }
      if (result[v] == BLUE
          && proposal[v] == idx
          && weights[i] >= weight_max) {
        v_max = v;
        weight_max = weights[i];
      }
    }
    if (v_max >= 0) {
      result[v_max] = min(idx, v_max);
      result[idx] = min(idx, v_max);
    }

    if (!has_unmatched_neighbors)
      result[idx] = idx;
  }
}

/*! \brief The colorize procedure. This procedure randomly marks unmarked
 * nodes with BLUE(-1) and RED(-2) and checks whether the node matching
 * process has finished.
 */
template<typename IdType>
bool Colorize(IdType * result_data, hiprandGenerator_t gen, int64_t num_nodes) {
  // initial done signal
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  CUDA_KERNEL_CALL(init_done_kernel, 1, 1, 0, thr_entry->stream);

  // generate color prop for each node
  float *prop;
  CUDA_CALL(hipMalloc(reinterpret_cast<void **>(&prop), num_nodes * sizeof(float)));
  CURAND_CALL(hiprandGenerateUniform(gen, prop, num_nodes));
  hipDeviceSynchronize();  // wait for random number generation finish since hiprand is async

  // call kernel
  auto num_threads = cuda::FindNumThreads(num_nodes);
  auto num_blocks = cuda::FindNumBlocks<'x'>(BLOCKS(num_nodes, num_threads));
  CUDA_KERNEL_CALL(colorize_kernel, num_blocks, num_threads, 0, thr_entry->stream,
                   prop, num_nodes, result_data);
  bool done_h = false;
  CUDA_CALL(hipMemcpyFromSymbol(&done_h, HIP_SYMBOL(done_d), sizeof(done_h), 0, hipMemcpyDeviceToHost));
  CUDA_CALL(hipFree(prop));
  return done_h;
}

/*! \brief Weighted neighbor matching procedure (GPU version).
 * This implementation is from `A GPU Algorithm for Greedy Graph Matching
 * <http://www.staff.science.uu.nl/~bisse101/Articles/match12.pdf>`__
 * 
 * This algorithm has three parts: colorize, propose and respond.
 * In colorize procedure, each unmarked node will be marked as BLUE or
 * RED randomly. If all nodes are marked, finish and return.
 * In propose procedure, each BLUE node will propose to the RED
 * neighbor with the largest weight (or randomly choose one if without weight).
 * If all its neighbors are marked, mark this node with its id.
 * In respond procedure, each RED node will respond to the BLUE neighbor
 * that has proposed to it and has the largest weight. If all neighbors
 * are marked, mark this node with its id. Else match this (BLUE, RED) node
 * pair and mark them with the smaller id between them.
 */
template <DLDeviceType XPU, typename FloatType, typename IdType>
void WeightedNeighborMatching(const aten::CSRMatrix &csr, const NDArray weight, IdArray result) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  if (!thr_entry->curand_gen) {
    uint64_t seed = dgl::RandomEngine::ThreadLocal()->RandInt(UINT64_MAX);
    CURAND_CALL(hiprandCreateGenerator(&thr_entry->curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(thr_entry->curand_gen, seed));
  }

  // create proposal tensor
  const int64_t num_nodes = result->shape[0];
  IdArray proposal = aten::Full(-1, num_nodes, sizeof(IdType) * 8, result->ctx);

  // get data ptrs
  IdType *indptr_data = static_cast<IdType*>(csr.indptr->data);
  IdType *indices_data = static_cast<IdType*>(csr.indices->data);
  IdType *result_data = static_cast<IdType*>(result->data);
  IdType *proposal_data = static_cast<IdType*>(proposal->data);
  FloatType *weight_data = static_cast<FloatType*>(weight->data);

  auto num_threads = cuda::FindNumThreads(num_nodes);
  auto num_blocks = cuda::FindNumBlocks<'x'>(BLOCKS(num_nodes, num_threads));
  while (!Colorize<IdType>(result_data, thr_entry->curand_gen, num_nodes)) {
    CUDA_KERNEL_CALL(weighted_propose_kernel, num_blocks, num_threads, 0, thr_entry->stream,
                     indptr_data, indices_data, weight_data, num_nodes, proposal_data, result_data);
    CUDA_KERNEL_CALL(weighted_respond_kernel, num_blocks, num_threads, 0, thr_entry->stream,
                     indptr_data, indices_data, weight_data, num_nodes, proposal_data, result_data);
  }
}
template void WeightedNeighborMatching<kDLGPU, float, int32_t>(
  const aten::CSRMatrix &csr, const NDArray weight, IdArray result);
template void WeightedNeighborMatching<kDLGPU, float, int64_t>(
  const aten::CSRMatrix &csr, const NDArray weight, IdArray result);
template void WeightedNeighborMatching<kDLGPU, double, int32_t>(
  const aten::CSRMatrix &csr, const NDArray weight, IdArray result);
template void WeightedNeighborMatching<kDLGPU, double, int64_t>(
  const aten::CSRMatrix &csr, const NDArray weight, IdArray result);

/*! \brief Unweighted neighbor matching procedure (GPU version).
 * Instead of directly sample neighbors, we assign each neighbor
 * with a random weight. We use random weight for 2 reasons:
 *  1. Random sample for each node in GPU is expensive. Although
 *     we can perform a global group-wise (neighborhood of each
 *     node as a group) random permutation as in CPU version,
 *     it still cost too much compared to directly using random weights.
 *  2. Graph is sparse, thus neighborhood of each node is small,
 *     which is suitable for GPU implementation.
 */
template <DLDeviceType XPU, typename IdType>
void NeighborMatching(const aten::CSRMatrix &csr, IdArray result) {
  const int64_t num_edges = csr.indices->shape[0];

  // generate random weights
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  if (!thr_entry->curand_gen) {
    uint64_t seed = dgl::RandomEngine::ThreadLocal()->RandInt(UINT64_MAX);
    CURAND_CALL(hiprandCreateGenerator(&thr_entry->curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(thr_entry->curand_gen, seed));
  }
  NDArray weight = NDArray::Empty(
    {num_edges}, DLDataType{kDLFloat, sizeof(float) * 8, 1}, result->ctx);
  float *weight_data = static_cast<float*>(weight->data);
  CURAND_CALL(hiprandGenerateUniform(thr_entry->curand_gen, weight_data, num_edges));
  hipDeviceSynchronize();

  WeightedNeighborMatching<XPU, float, IdType>(csr, weight, result);
}
template void NeighborMatching<kDLGPU, int32_t>(const aten::CSRMatrix &csr, IdArray result);
template void NeighborMatching<kDLGPU, int64_t>(const aten::CSRMatrix &csr, IdArray result);

}  // namespace impl
}  // namespace geometry
}  // namespace dgl
