/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/sddmm.cu
 * \brief SDDMM C APIs and definitions.
 */
#include <dgl/array.h>
#include "./sddmm.cuh"
#include "./functor.cuh"

namespace dgl {
namespace aten {

#define SWITCH_OP(op, Op, ...)                                      \
  do {                                                              \
    if ((op) == "add") {                                            \
      typedef cuda::binary::Add<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "sub") {                                     \
      typedef cuda::binary::Sub<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "mul") {                                     \
      typedef cuda::binary::Mul<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "div") {                                     \
      typedef cuda::binary::Div<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "copy_lhs") {                                \
      typedef cuda::binary::CopyLhs<DType> Op;                      \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "copy_rhs") {                                \
      typedef cuda::binary::CopyRhs<DType> Op;                      \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "dot") {                                     \
      typedef cuda::binary::Dot<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else {                                                        \
      LOG(FATAL) << "Unsupported SpMM/SDDMM binary operator: " << op;     \
    }                                                               \
  } while (0)

#define SWITCH_RHS(rhs_target, RhsTarget, ...)                        \
  do {                                                                \
    if ((rhs_target) == 0) {                                          \
      constexpr int RhsTarget = 0;                                    \
      { __VA_ARGS__ }                                                 \
    } else if ((rhs_target) == 1) {                                   \
      constexpr int RhsTarget = 1;                                    \
      { __VA_ARGS__ }                                                 \
    } else if ((rhs_target) == 2) {                                   \
      constexpr int RhsTarget = 2;                                    \
      { __VA_ARGS__ }                                                 \
    } else {                                                          \
      LOG(INFO) << "Invalid rhs target: " << (rhs_target);            \
    }                                                                 \
  } while (0)

#define SWITCH_TARGET(lhs_target, rhs_target, LhsTarget, RhsTarget, ...)\
  do {                                                                  \
    if ((lhs_target) == 0) {                                            \
      constexpr int LhsTarget = 0;                                      \
      SWITCH_RHS(rhs_target, RhsTarget, __VA_ARGS__);                   \
    } else if ((lhs_target) == 1) {                                     \
      constexpr int LhsTarget = 1;                                      \
      SWITCH_RHS(rhs_target, RhsTarget, __VA_ARGS__);                   \
    } else if ((lhs_target) == 2) {                                     \
      constexpr int LhsTarget = 2;                                      \
      SWITCH_RHS(rhs_target, RhsTarget, __VA_ARGS__);                   \
    } else {                                                            \
      LOG(INFO) << "Invalid lhs target: " << (lhs_target);              \
    }                                                                   \
  } while (0)

/*!
 * \brief CUDA implementation of g-SDDMM on Csr format.
 */
template <int XPU, typename IdType, int bits>
void SDDMMCsr(const std::string& op,
              const BcastOff& bcast,
              const CSRMatrix& csr,
              NDArray lhs,
              NDArray rhs,
              NDArray out,
              int lhs_target,
              int rhs_target) {
  SWITCH_BITS(bits, DType, {
    SWITCH_OP(op, Op, {
      SWITCH_TARGET(lhs_target, rhs_target, LhsTarget, RhsTarget, {
        cuda::SDDMMCsr<IdType, DType, Op, LhsTarget, RhsTarget>(bcast, csr, lhs, rhs, out);
      });
    });
  });
}

/*!
 * \brief CUDA implementation of g-SDDMM on Coo format.
 */
template <int XPU, typename IdType, int bits>
void SDDMMCoo(const std::string& op,
              const BcastOff& bcast,
              const COOMatrix& coo,
              NDArray lhs,
              NDArray rhs,
              NDArray out,
              int lhs_target,
              int rhs_target) {
  SWITCH_BITS(bits, DType, {
    SWITCH_OP(op, Op, {
      SWITCH_TARGET(lhs_target, rhs_target, LhsTarget, RhsTarget, {
        cuda::SDDMMCoo<IdType, DType, Op, LhsTarget, RhsTarget>(bcast, coo, lhs, rhs, out);
      });
    });
  });
}

template void SDDMMCsr<kDLGPU, int32_t, 16>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out,
    int lhs_target, int rhs_target);
template void SDDMMCsr<kDLGPU, int64_t, 16>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out,
    int lhs_target, int rhs_target);
template void SDDMMCsr<kDLGPU, int32_t, 32>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out,
    int lhs_target, int rhs_target);
template void SDDMMCsr<kDLGPU, int64_t, 32>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out,
    int lhs_target, int rhs_target);
template void SDDMMCsr<kDLGPU, int32_t, 64>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out,
    int lhs_target, int rhs_target);
template void SDDMMCsr<kDLGPU, int64_t, 64>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out,
    int lhs_target, int rhs_target);

template void SDDMMCoo<kDLGPU, int32_t, 16>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out,
    int lhs_target, int rhs_target);
template void SDDMMCoo<kDLGPU, int64_t, 16>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out,
    int lhs_target, int rhs_target);
template void SDDMMCoo<kDLGPU, int32_t, 32>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out,
    int lhs_target, int rhs_target);
template void SDDMMCoo<kDLGPU, int64_t, 32>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out,
    int lhs_target, int rhs_target);
template void SDDMMCoo<kDLGPU, int32_t, 64>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out,
    int lhs_target, int rhs_target);
template void SDDMMCoo<kDLGPU, int64_t, 64>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out,
    int lhs_target, int rhs_target);

}  // namespace aten
}  // namespace dgl
