#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/spmm.cu
 * \brief SPMM C APIs and definitions.
 */
#include <dgl/array.h>
#include "./spmm.cuh"
#include "./ge_spmm.cuh"
#include "./functor.cuh"
#include "../../runtime/cuda/cuda_common.h"

namespace dgl {

using namespace cuda;

namespace aten {
namespace {

/*! \brief Call cuBLAS geam API for transpose operation for float and double. */
template <typename DType>
hipblasStatus_t Xgeam(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const DType* alpha, const DType* A, int lda,
    const DType* beta, const DType* B, int ldb,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t Xgeam<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const float* alpha, const float* A, int lda,
    const float* beta, const float* B, int ldb,
    float* C, int ldc) {
  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <>
hipblasStatus_t Xgeam<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const double* alpha, const double* A, int lda,
    const double* beta, const double* B, int ldb,
    double* C, int ldc) {
  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

/* \brief IndexSelect operator kernel implementation.
 * \note duplicate of IndexSelectKernel defined in array_index_select.cu
 */
template <typename DType, typename IdType>
__global__ void _IndexSelectKernel(
    const DType* __restrict__ in,
    const IdType* __restrict__ idx,
    DType* __restrict__ out,
    int n, int m) {
  int i = blockIdx.x;
  for (int j = threadIdx.x; j < m; j += blockDim.x)
    out[i * m + j] = in[idx[i] * m + j];
}

/* \brief Transpose operator kernel implementation.
 * \note not efficient but it's not a bottleneck, used for float16 dtype.
 */
template <typename DType>
__global__ void _TransposeKernel(
    const DType* __restrict__ in,
    DType* __restrict__ out,
    int n, int m) {
  int i = blockIdx.x;
  for (int j = threadIdx.x; j < m; j += blockDim.x)
    out[i * m + j] = in[j * n + i];
}

/*
 * \brief Tranpose the input matrix.
 * \param row number of rows of input matrix.
 * \param col number of columns of input matrix.
 */
template <typename DType>
void _Transpose(const DType* in, DType* out,
                int row, int col) {
  DType alpha = 1., beta = 0.;
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  if (!thr_entry->cublas_handle)
    CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
  CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle, thr_entry->stream));
  CUBLAS_CALL(Xgeam<DType>(
      thr_entry->cublas_handle,
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      row, col,
      &alpha, in, col,
      &beta, nullptr, row,
      out, row));
}

/*
 * \brief Tranpose the input matrix for data type half.
 * \note cuBLAS has no geam API for half data type, fallback to our kernel.
 */
template <>
void _Transpose<half>(const half* in, half* out,
                      int row, int col) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = FindNumThreads(row);
  int nb = col;
  CUDA_KERNEL_CALL(_TransposeKernel, nb, nt, 0, thr_entry->stream, in, out, col, row);
}

/*
 * \brief
 */
template <typename DType, typename IdType>
__global__ void _IndexSelectKernel(const DType* array, const IdType* index,
                                   int64_t length, DType* out) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = array[index[tx]];
    tx += stride_x;
  }
}

/* \brief IndexSelect operator.
 * \note duplicate of IndexSelect defined in array_op.h but it can
 *    not be applied to float16 dtype.
 */
template<typename DType, typename IdType>
NDArray _IndexSelect(NDArray array, NDArray index) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  const DType* array_data = static_cast<DType*>(array->data);
  const IdType* idx_data = static_cast<IdType*>(index->data);
  const int64_t arr_len = array->shape[0];
  const int64_t len = index->shape[0];
  NDArray ret = NDArray::Empty({len}, array->dtype, array->ctx);
  if (len == 0)
    return ret;
  DType* ret_data = static_cast<DType*>(ret->data);
  const int nt = FindNumThreads(len);
  const int nb = (len + nt - 1) / nt;
  CUDA_KERNEL_CALL(_IndexSelectKernel, nb, nt, 0, thr_entry->stream,
      array_data, idx_data, len, ret_data);
  return ret;
}

}  // namespace

namespace cusparse {

#if CUDART_VERSION < 11000
template <typename DType>
hipsparseStatus_t Xcsrmm2(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const DType* alpha, const hipsparseMatDescr_t descrA,
    const DType* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const DType* B, int ldb, const DType* beta, DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPSPARSE_STATUS_EXECUTION_FAILED;
}

template <>
hipsparseStatus_t Xcsrmm2<float>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const float* alpha, const hipsparseMatDescr_t descrA,
    const float* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const float* B, int ldb, const float* beta, float* C, int ldc) {
  return hipsparseScsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <>
hipsparseStatus_t Xcsrmm2<double>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const double* alpha, const hipsparseMatDescr_t descrA,
    const double* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const double* B, int ldb, const double* beta, double* C, int ldc) {
  return hipsparseDcsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}
#endif

/*! Cusparse implementation of SpMM on Csr format. */
template <typename DType, typename IdType>
void CusparseCsrmm2(
    const DLContext& ctx,
    const CSRMatrix& csr,
    const DType* B_data, const DType* A_data,
    DType* C_data,
    int x_length) {
  // We use csrmm2 to perform following operation:
  // C = A x B, where A is a sparse matrix in csr format, B is the dense matrix for node
  // feature tensor. However, since cusparse only supports column-major, while our tensor
  // is stored in row-major, the actual computation is:
  // C = trans(A x trans(B)).
  // Currently, we use cublasXgeam to implement transposition and allocate intermediate
  // workspace memory for this.
  const int m = csr.num_rows;
  const int n = x_length;
  const int k = csr.num_cols;
  const int nnz = csr.indices->shape[0];
  const DType alpha = 1.0;
  const DType beta = 0.0;
  // device
  auto device = runtime::DeviceAPI::Get(ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, thr_entry->stream));
  // all one data array
  DType* valptr = nullptr;
  if (!A_data) {
    valptr = static_cast<DType*>(device->AllocWorkspace(ctx, nnz * sizeof(DType)));
    _Fill(valptr, nnz, static_cast<DType>(1.));
  }
#if CUDART_VERSION >= 11000
  hipsparseSpMatDescr_t matA;
  hipsparseDnMatDescr_t matB, matC;
  constexpr auto dtype = cuda_dtype<DType>::value;
  constexpr auto idtype = cusparse_idtype<IdType>::value;
  CUSPARSE_CALL(hipsparseCreateCsr(&matA,
      m, k, nnz,
      static_cast<IdType*>(csr.indptr->data),
      static_cast<IdType*>(csr.indices->data),
      const_cast<DType*>(valptr? valptr : A_data),
      idtype, idtype,
      HIPSPARSE_INDEX_BASE_ZERO, dtype));
  CUSPARSE_CALL(hipsparseCreateDnMat(&matB,
      k, n, n,
      const_cast<DType*>(B_data), dtype, HIPSPARSE_ORDER_ROW));
  CUSPARSE_CALL(hipsparseCreateDnMat(&matC,
      m, n, n,
      C_data, dtype, HIPSPARSE_ORDER_ROW));

  auto transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  auto transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  size_t workspace_size;
  CUSPARSE_CALL(hipsparseSpMM_bufferSize(
      thr_entry->cusparse_handle, transA, transB,
      &alpha, matA, matB, &beta, matC,
      dtype, HIPSPARSE_SPMM_CSR_ALG2,
      &workspace_size));
  void* workspace = device->AllocWorkspace(ctx, workspace_size);
  CUSPARSE_CALL(hipsparseSpMM(
      thr_entry->cusparse_handle, transA, transB,
      &alpha, matA, matB, &beta, matC,
      dtype, HIPSPARSE_SPMM_CSR_ALG2,
      workspace));
  device->FreeWorkspace(ctx, workspace);

  CUSPARSE_CALL(hipsparseDestroySpMat(matA));
  CUSPARSE_CALL(hipsparseDestroyDnMat(matB));
  CUSPARSE_CALL(hipsparseDestroyDnMat(matC));
#else
  // allocate matrix for temporary transposed output
  DType* trans_out = static_cast<DType*>(device->AllocWorkspace(ctx, m * n * sizeof(DType)));

  hipsparseMatDescr_t descr;
  CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
  CUSPARSE_CALL(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
  CUSPARSE_CALL(Xcsrmm2<DType>(
      thr_entry->cusparse_handle,
      HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_TRANSPOSE,
      m, n, k, nnz, &alpha,
      descr, (valptr)? valptr : A_data,
      static_cast<int32_t*>(csr.indptr->data),
      static_cast<int32_t*>(csr.indices->data),
      B_data, n, &beta, trans_out, m));
  CUSPARSE_CALL(hipsparseDestroyMatDescr(descr));
  // transpose the output matrix
  _Transpose(trans_out, C_data, n, m);
  device->FreeWorkspace(ctx, trans_out);
#endif
  if (valptr)
    device->FreeWorkspace(ctx, valptr);
}
}  // namespace cusparse

#define SWITCH_OP(op, Op, ...)                                      \
  do {                                                              \
    if ((op) == "add") {                                            \
      typedef cuda::binary::Add<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "sub") {                                     \
      typedef cuda::binary::Sub<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "mul") {                                     \
      typedef cuda::binary::Mul<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "div") {                                     \
      typedef cuda::binary::Div<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "copy_lhs") {                                \
      typedef cuda::binary::CopyLhs<DType> Op;                      \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "copy_rhs") {                                \
      typedef cuda::binary::CopyRhs<DType> Op;                      \
      { __VA_ARGS__ }                                               \
    } else {                                                        \
      LOG(FATAL) << "Unsupported SpMM binary operator: " << op;     \
    }                                                               \
  } while (0)

/*!
 * \brief Determine whether cusparse SpMM function is applicable.
 */
template <int bits, typename IdType>
inline bool cusparse_available() {
#if CUDART_VERSION < 11000
  if (std::is_same<IdType, int>::value)
    if (bits > 16)
      return true;
  return false;
#else
  if (bits == 16)
    return false;  // cusparse's SpMM on fp16 is slow, temporally disabled.
  return true;
#endif
}

/*!
 * \brief CUDA implementation of g-SpMM on Csr format.
 * \note use cusparse if the reduce operator is `sum` and there is
 *       no broadcast, use dgl's kernel in other cases.
 */
template <int XPU, typename IdType, int bits>
void SpMMCsr(const std::string& op, const std::string& reduce,
             const BcastOff& bcast,
             const CSRMatrix& csr,
             NDArray ufeat,
             NDArray efeat,
             NDArray out,
             std::vector<NDArray> out_aux) {
  int64_t feat_len = bcast.out_len;
  bool is_scalar_efeat = efeat.NumElements() == csr.indices->shape[0];
  bool use_efeat = op != "copy_lhs";

  if (reduce == "sum") {
    if (op == "copy_lhs" && cusparse_available<bits, IdType>()) {  // cusparse
      int64_t x_length = 1;
      for (int i = 1; i < ufeat->ndim; ++i)
        x_length *= ufeat->shape[i];
      SWITCH_BITS(bits, DType, {
        cusparse::CusparseCsrmm2<DType, IdType>(
            ufeat->ctx, csr,
            static_cast<DType*>(ufeat->data),
            nullptr,
            static_cast<DType*>(out->data),
            x_length);
      });
    } else if (op == "mul" && is_scalar_efeat && cusparse_available<bits, IdType>()) {  // cusparse
      int64_t x_length = 1;
      for (int i = 1; i < ufeat->ndim; ++i)
        x_length *= ufeat->shape[i];
      if (!IsNullArray(csr.data)) {
        SWITCH_BITS(bits, DType, {
          efeat = _IndexSelect<DType, IdType>(efeat, csr.data);
        });
      }
      SWITCH_BITS(bits, DType, {
        cusparse::CusparseCsrmm2<DType, IdType>(
            ufeat->ctx, csr,
            static_cast<DType*>(ufeat->data),
            static_cast<DType*>(efeat->data),
            static_cast<DType*>(out->data),
            x_length);
      });
    } else {  // general kernel
      SWITCH_BITS(bits, DType, {
        SWITCH_OP(op, Op, {
          cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Sum<IdType, DType> >(
              bcast, csr, ufeat, efeat, out, NullArray(), NullArray());
        });
      });
    }
  } else if (reduce == "max") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Max<IdType, DType> >(
            bcast, csr, ufeat, efeat, out, out_aux[0], out_aux[1]);
      });
    });
  } else if (reduce == "min") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Min<IdType, DType> >(
            bcast, csr, ufeat, efeat, out, out_aux[0], out_aux[1]);
      });
    });
  } else {
    LOG(FATAL) << "Not implemented";
  }
}


/*!
 * \brief CUDA implementation of g-SpMM on Coo format.
 */
template <int XPU, typename IdType, int bits>
void SpMMCoo(const std::string& op, const std::string& reduce,
             const BcastOff& bcast,
             const COOMatrix& coo,
             NDArray ufeat,
             NDArray efeat,
             NDArray out,
             std::vector<NDArray> out_aux) {
  if (reduce == "sum") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Sum<IdType, DType, true> > (
            bcast, coo, ufeat, efeat, out, NullArray(), NullArray());
      });
    });
  } else if (reduce == "max") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Max<IdType, DType, true> > (
            bcast, coo, ufeat, efeat, out, out_aux[0], out_aux[1]);
      });
    });
  }  else if (reduce == "min") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Min<IdType, DType, true> > (
            bcast, coo, ufeat, efeat, out, out_aux[0], out_aux[1]);
      });
    });
  } else {
    LOG(FATAL) << "Not implemented";
  }
}

template void SpMMCsr<kDLGPU, int32_t, 16>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, 16>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int32_t, 32>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, 32>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int32_t, 64>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, 64>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);

template void SpMMCoo<kDLGPU, int32_t, 16>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, 16>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int32_t, 32>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, 32>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int32_t, 64>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, 64>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);


}  // namespace aten
}  // namespace dgl
